#include "hip/hip_runtime.h"
// System includes
#include <assert.h>
#include <stdio.h>
#include<math.h>
#include <time.h>
#include <chrono>
#include <stdio.h>
#include <stdlib.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

#include ""

#include <math.h> 

//���������� �������
#define N 1024
//���������� ������
#define BL 200

//��������
#define A -1000
#define B 1000

//��������
#define EPS 1e-5

//��� ����������� �� GPU

__global__ void staticReverse(float* result_d, double step)
{
    //������� ����� �������
    float a = A + blockIdx.x*threadIdx.x*step;
    float b = A + (blockIdx.x*threadIdx.x+1)*step;
    
    //������� f(x) = log(8*x) - 9*x + 3

    float ak = a;
    float bk = b;

    do
    {
        if ((log(8*ak) - 9*ak + 3)*(log(8*bk) - 9*bk + 3)* 1.0 > 0)
            return;

        float xk = (ak + bk)* 1.0 / 2;
        float fxk = log(8*xk) - 9*xk + 3;

        //���� �� ���� � 0 ������
        if (fxk < 1e-05)
            break;

        if ((log(8*ak) - 9*ak + 3)* fxk * 1.0 < 0)
        {
            bk = xk;
        }
        else
        {
            ak = xk;
        }

    }
    while(bk-ak > EPS);

    float x = (ak + bk) * 1.0 / 2;

    if ((log(8*x) - 9*x + 3) * 1.0 < 1e-5)
    {
      //  printf("d[%d, %d] = %f \n", blockIdx.x, threadIdx.x, x);

        result_d[blockIdx.x*threadIdx.x] = x;
    }

}

int main(int argc, char* argv)
{
    size_t free, total;
    printf("\n");
    hipMemGetInfo(&free, &total);
    printf("%d KB free of total %d KB\n", free / 1024, total / 1024);

    //������� ������ ��� ����������
    const unsigned int n = N*BL;
    float* result = new float[n];

    for (unsigned int i = 0; i < n; i++) {
        result[i] = 0;
    }

    //��������� ���
    float step = fabs(A-B) * 1.0 / n;

    ////---���������� �� ����������---
    printf("[Reverse computing Using CUDA] - Starting...\n");

    hipStream_t stream;

    //// ��������� ������ �� ����������

    float* result_d;
    checkCudaErrors(hipMalloc(&result_d, n * sizeof(float)));


    //// ��������� ������ �� ���������� � ������������� ���� �������
    checkCudaErrors(hipMemcpy(result_d, result, n*sizeof(float), hipMemcpyHostToDevice));

    checkCudaErrors(hipDeviceSynchronize());

    //// �������� ������� � ������ ��� �������
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    checkCudaErrors(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));


    printf("Computing result using CUDA Kernel...\n");

    //// ������ ������ �������
    checkCudaErrors(hipEventRecord(start, stream));

    //// ���������� ���� �� ���������� � �������� ���������� ���� �������

    staticReverse << <BL, N >> > (result_d, step);


    checkCudaErrors(hipStreamSynchronize(stream));

    //// ������ ��������� �������
    checkCudaErrors(hipEventRecord(stop, stream));

    //// ������������� � �������� ������������ �������
    checkCudaErrors(hipEventSynchronize(stop));

    //// ������ � ����� ������������������

    float m_sec_total = 0.0f;
    checkCudaErrors(hipEventElapsedTime(&m_sec_total, start, stop));

    float mc_sec_total = m_sec_total * 1000;
    printf(
        "Time GPU = %.10f microsec\n",
        mc_sec_total);

    //// ����������� ����������� � GPU �� CPU
    checkCudaErrors(
        hipMemcpy(result, result_d, n * sizeof(unsigned int), hipMemcpyDeviceToHost));

    checkCudaErrors(hipStreamSynchronize(stream));


    ////---���������� �� ����������---

    
    float cpu_result[n];

    for (unsigned int i = 0; i < n; i++)
    {
        cpu_result[i] = 0;
    }

    // �������� ����� �������
    auto begin = std::chrono::high_resolution_clock::now();

    //������� f(x) = log(8*x) - 9*x + 3

    for (unsigned int i = 0; i < n; i++)
    {
        //������� ����� �������
        float a = A + i*step;
        float b = A + (i+1)*step;

        float ak = a;
        float bk = b;

        do
        {
            if ((log(8*ak) - 9*ak + 3)*(log(8*bk) - 9*bk + 3)* 1.0 > 0)
                break;

            float xk = (ak + bk)* 1.0 / 2;
            float fxk = log(8*xk) - 9*xk + 3;

            //���� �� ���� � 0 ������
            if (fxk < 1e-05)
                break;

            if ((log(8*ak) - 9*ak + 3)*fxk* 1.0 < 0)
            {
                bk = xk;
            }
            else
            {
                ak = xk;
            }

        }
        while(bk-ak > EPS);

        if ((log(8*ak) - 9*ak + 3)*(log(8*bk) - 9*bk + 3)* 1.0 <= 0)
           { 
                float x = (ak + bk) * 1.0 / 2;

                if ((log(8*x) - 9*x + 3) * 1.0 < 1e-5)
                {
                    //printf("d[%d] = %f \n", i, x);

                     cpu_result[i] = x;
                }
                    
           }
    }
    
    
    //// ������������� ������ � ������� ����� ����������
    auto end = std::chrono::high_resolution_clock::now();
    auto elapsed = std::chrono::duration_cast<std::chrono::nanoseconds>(end - begin);

    printf(
        "Time CPU = %.10f microsec\n",
        elapsed.count() * 1e-3);

    ////�������� ��������

    for (unsigned int i = 0; i < n; i++)
    {
        if (fabs(result[i] - cpu_result[i]) > 1e-05)
        {
           printf("d[%d] != r[%d] (%f, %f) \n", i, i, result[i], cpu_result[i]);
        }

        if (result[i] > 1e-05)
        {
           printf("GPU root (%d) == %f \n", i, result[i]);
        }

        if (cpu_result[i] > 1e-05)
        {
           printf("CPU root (%d) == %f \n", i, cpu_result[i]);
        }
    }

    //// ������������ ������
    checkCudaErrors(hipFree(result_d));
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));

    hipDeviceReset();

    return 0;
}
