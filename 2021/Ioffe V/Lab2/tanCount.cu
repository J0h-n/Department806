#include "hip/hip_runtime.h"
// System includes
#include <assert.h>
#include <stdio.h>
#include<math.h>
#include <time.h>
#include <chrono>
#include <stdio.h>
#include <stdlib.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

#include ""

//���������� �������
#define N 1024
//���������� ������
#define BL 97

//��� ����������� �� GPU

__global__ void staticReverse(long* d, long n)
{
    __shared__ long s[N];
    long global_t = threadIdx.x + 1024 * blockIdx.x;
    int t = threadIdx.x;
    
    if (global_t >= n)
        return;

    //int tr = N - t - 1;
    s[t] = d[global_t];
    __syncthreads();
    d[n - global_t - 1] = s[t];
}

int main(int argc, char* argv)
{
    const long n = 100000;
    long a[n], d[n];

    for (long i = 0; i < n; i++) {
        a[i] = i + 1;
        d[i] = 0;
    }

    ////---���������� �� ����������---
    printf("[Reverse computing Using CUDA] - Starting...\n");

    hipStream_t stream;

    //// ��������� ������ �� ����������



    long* d_d;
    checkCudaErrors(hipMalloc(&d_d, n * sizeof(long)));


    //// ��������� ������ �� ���������� � ������������� ���� �������
    checkCudaErrors(hipMemcpy(d_d, a, n*sizeof(long), hipMemcpyHostToDevice));

    checkCudaErrors(hipDeviceSynchronize());

    //// �������� ������� � ������ ��� �������
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    checkCudaErrors(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));


    printf("Computing result using CUDA Kernel...\n");

    //// ������ ������ �������
    checkCudaErrors(hipEventRecord(start, stream));

    //// ���������� ���� �� ���������� � �������� ���������� ���� �������

    staticReverse << <BL, N >> > (d_d, n);


    checkCudaErrors(hipStreamSynchronize(stream));

    //// ������ ��������� �������
    checkCudaErrors(hipEventRecord(stop, stream));

    //// ������������� � �������� ������������ �������
    checkCudaErrors(hipEventSynchronize(stop));

    //// ������ � ����� ������������������

    float m_sec_total = 0.0f;
    checkCudaErrors(hipEventElapsedTime(&m_sec_total, start, stop));

    float mc_sec_total = m_sec_total * 1000;
    printf(
        "Time GPU = %.10f microsec\n",
        mc_sec_total);

    //// ����������� ����������� � GPU �� CPU
    checkCudaErrors(
        hipMemcpy(d, d_d, n * sizeof(long), hipMemcpyDeviceToHost));

    checkCudaErrors(hipStreamSynchronize(stream));


    ////---���������� �� ����������---



    // �������� ����� �������
    auto begin = std::chrono::high_resolution_clock::now();

    long r[n];

    for (long i = 0; i < n; i++)
    {
        r[i] = a[n - i - 1];
    }
    
    //// ������������� ������ � ������� ����� ����������
    auto end = std::chrono::high_resolution_clock::now();
    auto elapsed = std::chrono::duration_cast<std::chrono::nanoseconds>(end - begin);

    printf(
        "Time CPU = %.10f microsec\n",
        elapsed.count() * 1e-3);

    ////�������� ��������

    for (long i = 0; i < n; i++)
    {
        if (d[i] != r[i])
        {
          /* printf("Error: d[%d]!=r[%d] (%d, %d) \n", i, i, d[i], r[i]);*/
        }
    }

    //// ������������ ������
    checkCudaErrors(hipFree(d_d));
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));

    hipDeviceReset();

    return 0;
}
