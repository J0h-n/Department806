#include "hip/hip_runtime.h"
// System includes
#include <assert.h>
#include <stdio.h>
#include<math.h>
#include <time.h>
#include <chrono>
#include <stdio.h>
#include <stdlib.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

#include ""

//���������� �������
#define N 1024
//���������� ������
#define BL 98

//��� ����������� �� GPU

__global__ void staticReverse(unsigned int* data_d, unsigned int* result_d, unsigned int n)
{
    __shared__ unsigned int s[N];
    unsigned int global_t = threadIdx.x + 1024 * blockIdx.x;
   // printf("%d", sizeof(double));
    int t = threadIdx.x;
    
    if (global_t >= n)
        return;

    //int tr = N - t - 1;
    s[t] = data_d[global_t];
    __syncthreads();
    result_d[n - global_t - 1] = s[t];
}

int main(int argc, char* argv)
{
    size_t free, total;
    printf("\n");
    hipMemGetInfo(&free, &total);
    printf("%d KB free of total %d KB\n", free / 1024, total / 1024);

    const unsigned int n = 100000;
    unsigned int *data = new unsigned int[n];
    unsigned int* result = new unsigned int[n];

    for (unsigned int i = 0; i < n; i++) {
        data[i] = i + 1;
        result[i] = 0;
    }

    ////---���������� �� ����������---
    printf("[Reverse computing Using CUDA] - Starting...\n");

    hipStream_t stream;

    //// ��������� ������ �� ����������



    unsigned int* data_d;
    checkCudaErrors(hipMalloc(&data_d, n * sizeof(unsigned int)));

    unsigned int* result_d;
    checkCudaErrors(hipMalloc(&result_d, n * sizeof(unsigned int)));


    //// ��������� ������ �� ���������� � ������������� ���� �������
    checkCudaErrors(hipMemcpy(data_d, data, n*sizeof(unsigned int), hipMemcpyHostToDevice));

    checkCudaErrors(hipDeviceSynchronize());

    //// �������� ������� � ������ ��� �������
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    checkCudaErrors(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));


    printf("Computing result using CUDA Kernel...\n");

    //// ������ ������ �������
    checkCudaErrors(hipEventRecord(start, stream));

    //// ���������� ���� �� ���������� � �������� ���������� ���� �������

    staticReverse << <BL, N >> > (data_d, result_d, n);


    checkCudaErrors(hipStreamSynchronize(stream));

    //// ������ ��������� �������
    checkCudaErrors(hipEventRecord(stop, stream));

    //// ������������� � �������� ������������ �������
    checkCudaErrors(hipEventSynchronize(stop));

    //// ������ � ����� ������������������

    float m_sec_total = 0.0f;
    checkCudaErrors(hipEventElapsedTime(&m_sec_total, start, stop));

    float mc_sec_total = m_sec_total * 1000;
    printf(
        "Time GPU = %.10f microsec\n",
        mc_sec_total);

    //// ����������� ����������� � GPU �� CPU
    checkCudaErrors(
        hipMemcpy(result, result_d, n * sizeof(unsigned int), hipMemcpyDeviceToHost));

    checkCudaErrors(hipStreamSynchronize(stream));


    ////---���������� �� ����������---



    // �������� ����� �������
    auto begin = std::chrono::high_resolution_clock::now();

    unsigned int cpu_result[n];

    for (unsigned int i = 0; i < n; i++)
    {
        cpu_result[i] = data[n - i - 1];
    }
    
    //// ������������� ������ � ������� ����� ����������
    auto end = std::chrono::high_resolution_clock::now();
    auto elapsed = std::chrono::duration_cast<std::chrono::nanoseconds>(end - begin);

    printf(
        "Time CPU = %.10f microsec\n",
        elapsed.count() * 1e-3);

    ////�������� ��������

    for (unsigned int i = 0; i < n; i++)
    {
        //if (result[i] != cpu_result[i])
        if(i % 11111 == 0)
        {
           printf("d[%d] == r[%d] (%d, %d) \n", i, i, result[i], cpu_result[i]);
        }
    }

    //// ������������ ������
    checkCudaErrors(hipFree(data_d));
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));

    hipDeviceReset();

    return 0;
}
