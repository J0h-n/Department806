#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>

#include <helper_string.h>

#include "SobelFilter_kernels.h"

// ������ ��������
hipTextureObject_t texObject;
//������ �� ����
static hipArray *array = NULL;


#define MAX(a,b) ((a > b) ? a : b)

#define MIN(a,b) ((a > b) ? b : a)

// ������� ��������� ������
#define checkCudaErrors(err)           __checkCudaErrors (err, __FILE__, __LINE__)

inline void __checkCudaErrors(hipError_t err, const char *file, const int line)
{
    if (hipSuccess != err)
    {
        fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",
                file, line, (int)err, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}


__device__ unsigned char
ComputeSobel(unsigned char ul, // upper left
             unsigned char um, // upper middle
             unsigned char ur, // upper right
             unsigned char ml, // middle left
             unsigned char mm, // middle (unused)
             unsigned char mr, // middle right
             unsigned char ll, // lower left
             unsigned char lm, // lower middle
             unsigned char lr // lower right
    )
{
    short Horz = ur + 2*mr + lr - ul - 2*ml - ll;
    short Vert = ul + 2*um + ur - ll - 2*lm - lr;
    short Sum = (short)round(sqrt((float)((int)Horz*(int)Horz +(int)Vert*(int)Vert)));

    if (Sum < 0)
    {
        return 0;
    }
    else if (Sum > 0xff)
    {
        return 0xff;
    }

    return (unsigned char) Sum;
}


//������ ���� - �������������� ����� �����������, ������ ����� - ��������� ������� � �����

__global__ void
SobelCopyImageNew(Pixel* pSobelOriginal, unsigned int Pitch,
    int w, int h, hipTextureObject_t tex)
{
    //����� ����������� ���� ��������� ������� ������� ����� �� ������ �������������� �����. �� ���� ��� ����� �� ������� height (������� ������)*������ �������� 
    unsigned char* pSobel =
        (unsigned char*)(((char*)pSobelOriginal) + blockIdx.x * Pitch);

    
    int i = threadIdx.x;
        //������������ ������� �� 0 �� 255
        pSobel[i] = MIN(MAX((tex2D<unsigned char>(tex, (float)i, (float)blockIdx.x)), 0.f), 255.f);
}

__global__ void
SobelTexNew(Pixel* pSobelOriginal, unsigned int Pitch,
    int w, int h, hipTextureObject_t tex)
{
    //����� ����������� ���� ��������� ������� ������� ����� �� ������ �������������� �����. �� ���� ��� ����� �� ������� height (������� ������)*������ �������� 
    unsigned char* pSobel =
        (unsigned char*)(((char*)pSobelOriginal) + blockIdx.x * Pitch);

    //����� ������� 3*3 ������ ������� �������
    int i = threadIdx.x;
    unsigned char pix00 = tex2D<unsigned char>(tex, (float)i - 1, (float)blockIdx.x - 1);
    unsigned char pix01 = tex2D<unsigned char>(tex, (float)i + 0, (float)blockIdx.x - 1);
    unsigned char pix02 = tex2D<unsigned char>(tex, (float)i + 1, (float)blockIdx.x - 1);
    unsigned char pix10 = tex2D<unsigned char>(tex, (float)i - 1, (float)blockIdx.x + 0);
    unsigned char pix11 = tex2D<unsigned char>(tex, (float)i + 0, (float)blockIdx.x + 0);
    unsigned char pix12 = tex2D<unsigned char>(tex, (float)i + 1, (float)blockIdx.x + 0);
    unsigned char pix20 = tex2D<unsigned char>(tex, (float)i - 1, (float)blockIdx.x + 1);
    unsigned char pix21 = tex2D<unsigned char>(tex, (float)i + 0, (float)blockIdx.x + 1);
    unsigned char pix22 = tex2D<unsigned char>(tex, (float)i + 1, (float)blockIdx.x + 1);
    //������ �������� ������ ������� � ������� ��������� ������
    pSobel[i] = ComputeSobel(pix00, pix01, pix02,
        pix10, pix11, pix12,
        pix20, pix21, pix22);

    
}

extern "C" void setupTexture(int iw, int ih, Pixel *data)
{
    hipChannelFormatDesc desc;

    desc = hipCreateChannelDesc<unsigned char>();

    checkCudaErrors(hipMallocArray(&array, &desc, iw, ih));
    checkCudaErrors(hipMemcpy2DToArray(array, 0, 0, data, iw * sizeof(Pixel), 
                                        iw * sizeof(Pixel), ih, hipMemcpyHostToDevice));

    hipResourceDesc            texRes;
    memset(&texRes,0,sizeof(hipResourceDesc));

    texRes.resType            = hipResourceTypeArray;
    texRes.res.array.array    = array;

    hipTextureDesc             texDescr;
    memset(&texDescr,0,sizeof(hipTextureDesc));

    texDescr.normalizedCoords = false;
    texDescr.filterMode       = hipFilterModePoint;
    texDescr.addressMode[0]   = hipAddressModeWrap;
    texDescr.readMode = hipReadModeElementType;

    //������������� ���������� ������

    checkCudaErrors(hipCreateTextureObject(&texObject, &texRes, &texDescr, NULL));

}

extern "C" void deleteTexture(void)
{
    checkCudaErrors(hipFreeArray(array));
    checkCudaErrors(hipDestroyTextureObject(texObject));
}

//� ����������� �� ������ �������������� ����������� ��� ����������
extern "C" void sobelFilter(Pixel *odata, int iw, int ih, enum SobelDisplayMode mode)
{
    switch (mode)
    {
        case SOBELDISPLAY_IMAGE:
           // SobelCopyImage<<<ih, 1>>>(odata, iw, iw, ih, texObject);
            SobelCopyImageNew << <ih, iw >> > (odata, iw, iw, ih, texObject);
            break;

        case SOBELDISPLAY_SOBELTEX:
           // SobelTex<<<ih, 1>>>(odata, iw, iw, ih, texObject);
            SobelTexNew << <ih, iw >> > (odata, iw, iw, ih, texObject);
            break;
    }
}
