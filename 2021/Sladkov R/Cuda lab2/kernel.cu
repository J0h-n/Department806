#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include ""
#include <stdio.h>
#include <cmath>
#include <omp.h>
#include <iostream>

#ifndef __HIPCC__ 
#define __HIPCC__
#endif
#include <hip/device_functions.h>

#define N 100000

__global__ void kernel(int* out) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int answ = 1;

	for (int i = 2; i < idx; i++) {
		if (idx % i == 0) {
			answ += i;
		}
	}

	if (answ == idx) { 
		out[idx] = answ;
	}
	else
	{
		out[idx] = NULL;
	}
}

int main()
{

	hipEvent_t start, stop;
	float gpuTime = 0.0f;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	int* out = new int[N];
	int* dev;

	hipMalloc((void**)&dev, N * sizeof(float));
	hipDeviceSynchronize();

	dim3 dimThreads(N / 8, 1);
	dim3 dimBlocks(N / dimThreads.x, 1);

	kernel << <dimBlocks, dimThreads >> > (dev);

	hipMemcpy(out, dev, N * sizeof(float), hipMemcpyDeviceToHost);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&gpuTime, start, stop);
	printf("N = %d\n\nGPU compute time: %.10f milliseconds\n", N, gpuTime);

	//for (int i = 0; i < N; i++)
		//if(out[i]) printf("%d ", out[i]);

	hipFree(dev);
	hipDeviceReset();

	double start2;
	double end2;
	start2 = omp_get_wtime();

	float* a = new float[N];

	for (int i = 0; i < N; i++) {
		int answ = 1;

		for (int j = 2; i < i; j++) {
			if (i % j == 0) {
				answ += j;
			}
		}

		if (answ == i) a[i] == answ;
	}

	end2 = omp_get_wtime();

	printf("\n\nCPU compute time: %f milliseconds\n", (end2 - start2) * 1000);

	return 0;
}
